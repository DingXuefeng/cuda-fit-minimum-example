#include "hip/hip_runtime.h"
#include "TMinuit.h"
#include <functional>
#include <iostream>

class FCN {
  public:
    virtual void evaluate(int&,double*,double&,double [],int) = 0;
    static void setCurrentFCN(FCN * fcn) { current = fcn; }
    static void wrapper(int& npar,double* deriv,double& f,double par[],int flag) {
      if(current) current->evaluate(npar,deriv,f,par,flag);
    }
  private:
    static FCN *current;
};

FCN *FCN::current = nullptr;

class FCN_serial: public FCN {
  public:
    FCN_serial(std::vector<double> _xs,std::vector<int> _data) : xs(_xs),data(_data) { }
    void evaluate(int& npar,double*,double& f,double par[],int) final {
      std::vector<double> result;
      for(size_t i = 0; i<data.size();++i) {
        result.push_back(this->get_LL(xs[i],data[i],par));
      }
      double sum = 0;
      for(size_t i = 0; i<data.size();++i) {
        sum += result[i];
      }
      f = sum;
    }
  private:
    double get_T(double x,double par[]) {
      return exp(par[0])+par[1]*x;
    }
    double get_LL(double x,int M,double par[]) {
      double T = this->get_T(x,par);
      return -M*log(T)+T+lgamma(M+1);
    }
    std::vector<double> xs;
    std::vector<int> data;
};

#include <thrust/transform_reduce.h>
#include <thrust/device_vector.h>
class FCN_cuda: public FCN {
  public:
    FCN_cuda(std::vector<double> _xs,std::vector<int> _data) : xs(_xs),data(_data) { }
    void evaluate(int& npar,double*,double& f,double par[],int) final {
      serial(par);
      f = parallel(par);
    }
  private:
    void serial(double par[]) {
      par[0] = exp(par[0]);
    }
    double parallel(double par[]) {
      thrust::device_vector<double> dev_x = xs;
      thrust::device_vector<int> dev_data = data;
      cuda_evalute_helper op(par[0],par[1]);
      auto begin = thrust::make_zip_iterator(thrust::make_tuple(dev_x.begin(), dev_data.begin()));
      auto end = thrust::make_zip_iterator(thrust::make_tuple(dev_x.end(), dev_data.end()));
      return thrust::transform_reduce(begin,end,op,0.,thrust::plus<double>());
    }
    std::vector<double> xs;
    std::vector<int> data;
  public:
    struct cuda_evalute_helper {
      const double p0_exp;
      const double p1;
      cuda_evalute_helper(double _p0_exp,double _p1): p0_exp(_p0_exp),p1(_p1) { }
     __device__
        double operator()(thrust::tuple<double,int> tuple) const {
          double x = thrust::get<0>(tuple);
          double M = thrust::get<1>(tuple);
          double T = p0_exp+p1*x;
          return -M*log(T)+T+lgamma(M+1);
        }
    };
};

int main() {
  std::vector<double> xs = {1,2,3,4,5,6,7};
  std::vector<int> data = {10,11,9,8,12,10,10};

  TMinuit fitter(2);
  fitter.DefineParameter(0,"p0",log(10),log(10)/100,-10,10);
  fitter.DefineParameter(1,"p1",0,1,-5,5);

  std::map<std::string,FCN*> fcns = {
    {"serial", new FCN_serial(xs,data)},
    {"cuda", new FCN_cuda(xs,data)}
  };

  for(auto fcn : fcns) {
    FCN::setCurrentFCN(fcn.second);
    fitter.SetFCN(FCN::wrapper);
    fitter.Migrad();
    double p0,p0e,p1,p1e;
    fitter.GetParameter(0,p0,p0e);
    fitter.GetParameter(1,p1,p1e);
    std::cout<<fcn.first<<" p0: "<<p0<<" ± "<<p0e<<"; p1: "<<p1<<" ± "<<p1e<<std::endl;
  }
  return 0;
}
